#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <time.h>

#define N 16         // size of array

__global__ void add(double *a, double *c) {
	int tid = blockIdx.x *  blockDim.x + threadIdx.x;
        if(tid < N){
          c[tid] = sin(a[tid]);
        }
}

int main(int argc, char *argv[])  {
	int T = 10, B = 1;            // threads per block and blocks per grid
	double a[N], c[N];
	double *dev_a, *dev_c;
	//double size = (double)N;

	printf("Size of array = %d\n", N);
	do {
		printf("Enter number of threads per block: ");
		scanf("%d",&T);
		printf("\nEnter nuumber of blocks per grid: ");
		scanf("%d",&B);
		if (T * B != N) printf("Error T x B != N, try again");
	} while (T * B != N);

	hipEvent_t start, stop;     // using cuda events to measure time
	float elapsed_time_ms;       // which is applicable for asynchronous code also

	hipMalloc((void**)&dev_a,N * sizeof(double));
	//cudaMalloc((void**)&dev_b,N * sizeof(int));
	hipMalloc((void**)&dev_c,N * sizeof(double));

	for(int i = 0; i < N; i++) {    // load array with some numbers
		a[i] = (double)i;
	}

	hipMemcpy(dev_a, a , N*sizeof(double),hipMemcpyHostToDevice);
	hipMemcpy(dev_c, c , N*sizeof(double),hipMemcpyHostToDevice);

	hipEventCreate( &start );     // instrument code to measure start time
	hipEventCreate( &stop );
	hipEventRecord( start, 0 );

	add<<<B,T>>>(dev_a,dev_c);

	hipMemcpy(c,dev_c,N*sizeof(double),hipMemcpyDeviceToHost);

	hipEventRecord( stop, 0 );     // instrument code to measue end time
	hipEventSynchronize( stop );
	hipEventElapsedTime( &elapsed_time_ms, start, stop );

	for(int i = 0; i < N; i++) {
		printf("sin(%lf) = %lf\n",a[i], c[i]);
	}

	printf("Time to calculate results: %f ms.\n", elapsed_time_ms);  // print out execution time

	// clean up
	hipFree(dev_a);
	hipFree(dev_c);

	hipEventDestroy(start);
	hipEventDestroy(stop);

	return 0;
}
