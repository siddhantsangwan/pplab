#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <time.h>

#define N 512         // size of array

__global__ void add(int *a,int *b, int *c, int *scalar) {
	int tid = blockIdx.x *  blockDim.x + threadIdx.x;
		int s = *scalar;
        if(tid < N){
          c[tid] = s * a[tid] + b[tid];
        }
}

int main(int argc, char *argv[])  {
	int T = 10, B = 1;            // threads per block and blocks per grid
	int a[N],b[N],c[N], *scalar;

	scalar = (int*)malloc(sizeof(int));
	int *dev_a, *dev_b, *dev_c, *dev_scalar;

	printf("Enter the scalar:\n");
	scanf("%d", scalar);

	printf("Size of array = %d\n", N);
	do {
		printf("Enter number of threads per block: ");
		scanf("%d",&T);
		printf("\nEnter number of blocks per grid: ");
		scanf("%d",&B);
		if (T * B != N) printf("Error T x B != N, try again");
	} while (T * B != N);

	hipEvent_t start, stop;     // using cuda events to measure time
	float elapsed_time_ms;       // which is applicable for asynchronous code also

	hipMalloc((void**)&dev_a,N * sizeof(int));
	hipMalloc((void**)&dev_b,N * sizeof(int));
	hipMalloc((void**)&dev_c,N * sizeof(int));
	hipMalloc((void**)&dev_scalar, sizeof(int));

	for(int i=0;i<N;i++) {    // load arrays with some numbers
		a[i] = i;
		b[i] = i*1;
	}

	hipMemcpy(dev_a, a, N*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, N*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_c, c, N*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_scalar, scalar, sizeof(int), hipMemcpyHostToDevice);

	hipEventCreate( &start );     // instrument code to measure start time
	hipEventCreate( &stop );
	hipEventRecord( start, 0 );

	add<<<B,T>>>(dev_a, dev_b, dev_c, dev_scalar);

	hipMemcpy(c,dev_c,N*sizeof(int),hipMemcpyDeviceToHost);

	hipEventRecord( stop, 0 );     // instrument code to measue end time
	hipEventSynchronize( stop );
	hipEventElapsedTime( &elapsed_time_ms, start, stop );

	for(int i=0;i<N;i++) {
		printf("(%d * %d) + %d = %d\n", *scalar, a[i], b[i], c[i]);
	}

	printf("Time to calculate results: %f ms.\n", elapsed_time_ms);  // print out execution time

	// clean up
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);

	hipEventDestroy(start);
	hipEventDestroy(stop);

	return 0;
}
